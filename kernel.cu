#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
using namespace std;
#define N (1024 * 1024)

__global__ void kernel(float* data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float x = 2 * 3.1415926 * (float)idx / (float)N;
	data[idx] = sinf(sqrtf(x));
}
int main(int argc, char* argv[])
{
	// Начало измерения времени
	clock_t start_time = clock();

	float* a = (float*)malloc(N * sizeof(float));
	float* dev = nullptr;

	// выделить память на GPU
	hipMalloc((void**)&dev, N * sizeof(float));

	// Получаем информацию об устройстве
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);  // Используем первый GPU
	int numMultiprocessors = devProp.multiProcessorCount;
	int maxThreadsPerBlock = devProp.maxThreadsPerBlock;

	// Вычисление количества блоков
	int blocksPerMultiprocessor = maxThreadsPerBlock / maxThreadsPerBlock;
	int numBlocks = numMultiprocessors * blocksPerMultiprocessor;

	// Конфигурация запуска
	kernel << <dim3(blocksPerMultiprocessor), dim3(maxThreadsPerBlock) >> > (dev);

	// конфигурация запуска N нитей
	//kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);
	// скопировать результаты в память CPU
	hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
	// освободить выделенную память
	hipFree(dev);
	
	//for (int idx = 0; idx < N; idx++)
	//	printf("a[%d] = %.5f\n", idx, a[idx]);
	free(a);
	
	int deviceCount;
	//hipDeviceProp_t devProp;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);
	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);
		printf("Device %d\n", device);
		printf("Compute capability : %d.%d\n", devProp.major, devProp.minor);
		printf("Name : %s\n", devProp.name);
		printf("Total Global Memory : %u\n", devProp.totalGlobalMem);
		printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
		printf("Registers per block : %d\n", devProp.regsPerBlock);
		printf("Warp size : %d\n", devProp.warpSize);
		printf("Max threads per block : %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory : %d\n", devProp.totalConstMem);
		printf("MultiProcessor count : %d\n", devProp.multiProcessorCount);
		printf("Kernel Exec Timeout Enabled : %d\n", numBlocks);
		printf("Kernel Exec Timeout Enabled : %d\n", devProp.kernelExecTimeoutEnabled);
	}

	// Конец измерения времени
	clock_t end_time = clock();

	// Разница во времени
	double elapsed_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

	printf("Время выполнения: %f секунд\n", elapsed_time);

	return 0;
}